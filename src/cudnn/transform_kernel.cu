
#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

void Transform::map(void)
{
  if (src_layout == dst_layout)
    return;
  //TODO: for now the output and input share the same instance
  checkCUDNN(hipdnnCreateTensorDescriptor(&srcTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&dstTensor));

  if (src_layout == HIPDNN_TENSOR_NCHW)
  {
    N = inputs[0].dim[0];
    C = inputs[0].dim[1];
    H = inputs[0].dim[2];
    W = inputs[0].dim[3];
  }
  else if (src_layout == HIPDNN_TENSOR_NHWC)
  {
    N = inputs[0].dim[0];
    H = inputs[0].dim[1];
    W = inputs[0].dim[2];
    C = inputs[0].dim[3];
  }
  // set descriptors
  checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensor, src_layout,
    HIPDNN_DATA_FLOAT, N, C, H, W));
  checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensor, dst_layout,
    HIPDNN_DATA_FLOAT, N, C, H, W));

  // allocate tensors
  size_t outputSize = sizeof(DATATYPE) * N * C * H * W;
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputSize));
}  

void Transform::unmap(void)
{
  checkCUDNN(hipdnnDestroyTensorDescriptor(srcTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(dstTensor));
  // free tensors
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void Transform::forward(bool block)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  checkCUDNN(cudnnTransformTensor(
    model->dnn, &alpha, srcTensor, inputs[0].data_ptr,
    &beta, dstTensor, outputs[0].data_ptr));

  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_transform_cost(Transform* transform)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;

  // set descriptors
  checkCUDNN(hipdnnSetTensor4dDescriptor(transform->srcTensor, transform->src_layout,
    HIPDNN_DATA_FLOAT, transform->N, transform->C, transform->H, transform->W));
  checkCUDNN(hipdnnSetTensor4dDescriptor(transform->dstTensor, transform->dst_layout,
    HIPDNN_DATA_FLOAT, transform->N, transform->C, transform->H, transform->W));

  

  checkCUDA(hipDeviceSynchronize());
  for (int i = 0; i < WARMUP_TIMES + REPEAT_TIMES; i++) {
    if (i == WARMUP_TIMES) {
      checkCUDA(hipEventRecord(startEvent));
    }

    checkCUDNN(cudnnTransformTensor(
      dnn, &alpha, transform->srcTensor, transform->inputs[0].data_ptr,
      &beta, transform->dstTensor, transform->outputs[0].data_ptr));
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  transform->runtime = milliseconds / REPEAT_TIMES;
  if (print_cost)
    printf("  measure[Transform]: s(%d %d %d %d) layout(%d -> %d) cost(%.4lf)\n",
      transform->N, transform->C, transform->H, transform->W,
      transform->src_layout, transform->dst_layout, transform->runtime
    );
}
